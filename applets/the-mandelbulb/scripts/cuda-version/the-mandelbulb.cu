#include "hip/hip_runtime.h"
#include "lodepng.h"
#include <iostream>
#include <cmath>



__device__ float distance_estimator(float* pos, float cx, float cy, float cz, float* color)
{
	float z[3] = {pos[0], pos[1], pos[2]};
	
	float r = sqrtf(z[0]*z[0] + z[1]*z[1] + z[2]*z[2]);;
	float dr = 1.0f;
	
	color[0] = 1.0f;
	color[1] = 1.0f;
	color[2] = 1.0f;
	
	float color_scale = .5f;
	
	
	
	//32 max iterations.
	for (int i = 0; i < 32; i++)
	{
		if (r > 16.0f)
		{
			break;
		}
		
		float theta = 8.0 * acosf(z[2] / r);
		
		float phi = 8.0 * atan2f(z[1], z[0]);
		
		dr = powf(r, 7.0f) * 8.0f * dr + 1.0f;
		
		float factor = powf(r, 8.0f);
		
		z[0] = factor * cosf(phi) * sinf(theta) + cx;
		z[1] = factor * sinf(phi) * sinf(theta) + cy;
		z[2] = factor * cosf(theta) + cz;
		
		r = sqrtf(z[0]*z[0] + z[1]*z[1] + z[2]*z[2]);
		
		color[0] = (1.0f - color_scale) * color[0] + color_scale * fabsf(z[0] / r);
		color[1] = (1.0f - color_scale) * color[1] + color_scale * fabsf(z[1] / r);
		color[2] = (1.0f - color_scale) * color[2] + color_scale * fabsf(z[2] / r);
		
		color_scale *= .5f;
	}
	
	
	
	float max_color_component = fmaxf(fmaxf(color[0], color[1]), color[2]);
	
	color[0] /= max_color_component;
	color[1] /= max_color_component;
	color[2] /= max_color_component;
	
	return .5f * logf(r) * r / dr;
}



__device__
void raymarch(float x, float y, float cx, float cy, float cz, float* color, float image_size)
{
	//The center of the image plane is (.052368, 1.588541, 1.400874), the right vector is (-.380503, .019914, 0), and the up vector is (-.012847, -.245477, .291128).
	
	float start_pos[3] = {.052368f - .380503f * x - .012847f * y, 1.588541f + .019914f * x - .245477f * y, 1.400874f + .291128f * y};
	
	
	
	//The camera pos is (.0828, 2.17, 1.8925).
	float ray_direction_vec_source[3] = {start_pos[0] - .0828f, start_pos[1] - 2.17f, start_pos[2] - 1.8925f};
	
	float magnitude = sqrtf(ray_direction_vec_source[0]*ray_direction_vec_source[0] + ray_direction_vec_source[1]*ray_direction_vec_source[1] + ray_direction_vec_source[2]*ray_direction_vec_source[2]);
	
	ray_direction_vec_source[0] /= magnitude;
	ray_direction_vec_source[1] /= magnitude;
	ray_direction_vec_source[2] /= magnitude;
	
	float ray_direction_vec[3] = {ray_direction_vec_source[0] * .9f, ray_direction_vec_source[1] * .9f, ray_direction_vec_source[2] * .9f};
	
	
	
	float garbage_color[3] = {0.0f, 0.0f, 0.0f};
	
	
	
	float epsilon = 0.0f;
	
	float t = 0.0f;
	
	float last_distance = 1000.0f;
	
	bool slowed_down = false;
	
	int iteration;
	
	
	
	//512 max marches.
	for (iteration = 0; iteration < 512; iteration++)
	{
		float pos[3] = {start_pos[0] + t * ray_direction_vec[0], start_pos[1] + t * ray_direction_vec[1], start_pos[2] + t * ray_direction_vec[2]};
		
		float distance = min(distance_estimator(pos, cx, cy, cz, color), last_distance);
		last_distance = distance;
		
		if (distance / image_size * 0.5f > epsilon)
		{
			epsilon = distance / image_size * 0.5f;
		}
		
		
		
		if (distance < epsilon)
		{
			//Compute shading.
			
			float pos_x[3] = {pos[0] + .000001f, pos[1], pos[2]};
			float pos_y[3] = {pos[0], pos[1] + .000001f, pos[2]};
			float pos_z[3] = {pos[0], pos[1], pos[2] + .000001f};
			
			float base = distance_estimator(pos, cx, cy, cz, garbage_color);
			
			float x_step = distance_estimator(pos_x, cx, cy, cz, garbage_color);
			float y_step = distance_estimator(pos_y, cx, cy, cz, garbage_color);
			float z_step = distance_estimator(pos_z, cx, cy, cz, garbage_color);
			
			float surface_normal[3] = {x_step - base, y_step - base, z_step - base};
			
			magnitude = sqrtf(surface_normal[0]*surface_normal[0] + surface_normal[1]*surface_normal[1] + surface_normal[2]*surface_normal[2]);
			
			surface_normal[0] /= magnitude;
			surface_normal[1] /= magnitude;
			surface_normal[2] /= magnitude;
			
			
			
			float light_direction[3] = {-pos[0], -pos[1], 5.0f - pos[2]};
			
			magnitude = sqrtf(light_direction[0]*light_direction[0] + light_direction[1]*light_direction[1] + light_direction[2]*light_direction[2]);
			
			light_direction[0] /= magnitude;
			light_direction[1] /= magnitude;
			light_direction[2] /= magnitude;
			
			
			
			float dot_product = surface_normal[0]*light_direction[0] + surface_normal[1]*light_direction[1] + surface_normal[2]*light_direction[2];
			
			float light_scale = 2.0f * fmaxf(dot_product, -.25f * dot_product) * fmaxf(1.0f - iteration / 512.0f, 0.0f);
			
			
			
			float distance_from_camera = sqrtf((pos[0] - .0828f)*(pos[0] - .0828f) + (pos[1] - 2.17f)*(pos[1] - 2.17f) + (pos[2] - 1.8925f)*(pos[2] - 1.8925f));
			
			float fog_scale = exp(-distance_from_camera * .2f);
			
			
			
			color[0] = color[0] * light_scale * fog_scale;
			color[1] = color[1] * light_scale * fog_scale;
			color[2] = color[2] * light_scale * fog_scale;
			
			break;
		}
		
		
		
		else if (last_distance / distance > .9999f && !slowed_down)
		{
			ray_direction_vec[0] = ray_direction_vec_source[0] * .125f;
			ray_direction_vec[1] = ray_direction_vec_source[1] * .125f;
			ray_direction_vec[2] = ray_direction_vec_source[2] * .125f;
			
			slowed_down = true;
		}
		
		else if (last_distance / distance <= .9999f && slowed_down)
		{
			ray_direction_vec[0] = ray_direction_vec_source[0] * .9f;
			ray_direction_vec[1] = ray_direction_vec_source[1] * .9f;
			ray_direction_vec[2] = ray_direction_vec_source[2] * .9f;
			
			slowed_down = false;
		}
		
		
		
		//Clip distance
		else if (t > 100.0)
		{
			color[0] = 0.0f;
			color[1] = 0.0f;
			color[2] = 0.0f;
			
			break;
		}
		
		
		
		t += distance;
	}
	
	
	
	if (iteration == 512)
	{
		color[0] = 0.0f;
		color[1] = 0.0f;
		color[2] = 0.0f;
	}
}



__global__
void generate_mandelbulb(unsigned char* image, int sector_row, int sector_col, float cx, float cy, float cz, int image_size)
{
	float color[3];
	float total_color[3] = {0.0, 0.0, 0.0};
	
	int index = 4 * (256 * blockIdx.x + threadIdx.x);
	
	
	
	//Why yes, that 255 is disturbing. No, I don't have any idea why it doesn't make a gap.
	
	float y = 1.0f - 2.0f * (float) (255 * sector_row + blockIdx.x) / (float) image_size;
	
	float x = 2.0f * (float) (255 * sector_col + threadIdx.x) / (float) image_size - 1.0f;
	
	float step = .5f / (float) image_size;
	
	
	
	raymarch(x - step, y - step, cx, cy, cz, color, (float) image_size);
	
	total_color[0] += color[0];
	total_color[1] += color[1];
	total_color[2] += color[2];
	
	
	
	raymarch(x + step, y - step, cx, cy, cz, color, (float) image_size);
	
	total_color[0] += color[0];
	total_color[1] += color[1];
	total_color[2] += color[2];
	
	
	
	raymarch(x - step, y + step, cx, cy, cz, color, (float) image_size);
	
	total_color[0] += color[0];
	total_color[1] += color[1];
	total_color[2] += color[2];
	
	
	
	raymarch(x + step, y + step, cx, cy, cz, color, (float) image_size);
	
	total_color[0] += color[0];
	total_color[1] += color[1];
	total_color[2] += color[2];
	
	
	
	image[index] = min(total_color[0] / 4.0f * 255.0f, 255.0f);
	image[index + 1] = min(total_color[1] / 4.0f * 255.0f, 255.0f);
	image[index + 2] = min(total_color[2] / 4.0f * 255.0f, 255.0f);
	image[index + 3] = 255;
}



void encode_image(const char* filename, std::vector<unsigned char>& image, unsigned width, unsigned height)
{
	//Encode the image.
	unsigned error = lodepng::encode(filename, image, width, height);

	//If there's an error, display it.
	if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;
}



int main(void)
{
	int image_size;
	
	int starting_frame;
	
	
	
	std::cout << "Image size: 2^";
	std::cin >> image_size;
	
	image_size = pow(2, image_size);
	
	std::cout << "Starting frame: ";
	std::cin >> starting_frame;
	
	
	
	float cx, cy, cz;
	
	unsigned char* d_image;
	
	char filename[9] = "0000.png";
	
	hipMallocManaged(&d_image, 4 * 256 * 256 * sizeof(unsigned char));
	
	int num_sectors = image_size / 256;
	
	std::vector<unsigned char> image_vector;
	image_vector.resize((image_size + num_sectors) * (image_size + num_sectors) * 4);
	
	for (int i = 0; i < (image_size + num_sectors) * (image_size + num_sectors); i++)
	{
		image_vector[4 * i] = 0;
		image_vector[4 * i + 1] = 0;
		image_vector[4 * i + 2] = 0;
		image_vector[4 * i + 3] = 255;
	}
	
	
	
	filename[3] = (starting_frame % 10) + 48;
	filename[2] = ((starting_frame / 10) % 10) + 48;
	filename[1] = ((starting_frame / 100) % 10) + 48;
	filename[0] = ((starting_frame / 1000) % 10) + 48;
	
	
	
	for (int frame = starting_frame; frame < 6000; frame++)
	{
		cx = .5f * (cosf(6.283185f * (float) frame / 6000.0f) + sinf(5.0f * 6.283185f * (float) frame / 6000.0f));
		cy = .5f * (cosf(2 * 6.283185f * (float) frame / 6000.0f) + sinf(7.0f * 6.283185f * (float) frame / 6000.0f));
		cz = .5f * (cosf(3 * 6.283185f * (float) frame / 6000.0f) + sinf(11.0f * 6.283185f * (float) frame / 6000.0f));
		
		for (int i = 0; i < num_sectors; i++)
		{
			for (int j = 0; j < num_sectors; j++)
			{
				std::cout << "Frame " << frame << ": sector " << num_sectors * i + j + 1 << " of " << num_sectors * num_sectors << std::endl;
				
				generate_mandelbulb<<<256, 256>>>(d_image, i, j, cx, cy, cz, image_size);
				
				hipDeviceSynchronize();
				
				
				
				for (int k = 0; k < 256; k++)
				{
					for (int l = 0; l < 256; l++)
					{
						int big_index = 4 * ((image_size + num_sectors) * (256 * i + k) + (256 * j + l));
						int small_index = 4 * (256 * k + l);
						
						image_vector[big_index] = d_image[small_index];
						image_vector[big_index + 1] = d_image[small_index + 1];
						image_vector[big_index + 2] = d_image[small_index + 2];
						image_vector[big_index + 3] = 255;
					}
				}
			}
		}
		
		std::cout << std::endl;
		
		
		
		encode_image(filename, image_vector, image_size + num_sectors, image_size + num_sectors);
		
		
		
		filename[3]++;
		
		if (filename[3] == 58)
		{
			filename[3] = 48;
			
			filename[2]++;
			
			if (filename[2] == 58)
			{
				filename[2] = 48;
				
				filename[1]++;
				
				if (filename[1] == 58)
				{
					filename[1] = 48;
					
					filename[0]++;
				}
			}
		}
	}
	
	
	
	hipFree(d_image);
	
	return 0;
}